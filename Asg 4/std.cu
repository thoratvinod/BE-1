#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
using namespace std;

__global__ void sum(float* input,int count)
{
	int tid = threadIdx.x;

	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{
			int fst = tid * step_size * 2;
			int snd = fst + step_size;
			if(fst==count-1)
		    	input[0]+=input[fst];
		    else  
				input[fst] += input[snd];
		}
		step_size *= 2; 
		number_of_threads /= 2;
	}
}

__global__ void ss(float* input,float *output,float mean)
{
   int index = threadIdx.x; 
   output[index] = (input[index]-mean)*(input[index]-mean);
}

int main()
{
	int count = 10;
  int size = count * sizeof(float);
	float h[count],sm[count]; 
    
  srand(time(NULL));
	for(int i=0;i<count;i++)
    {
	     	h[i] = rand()%50;
        cout<<h[i]<<",";
    }
	
  float *d,*output;
	
	hipMalloc(&d, size);
  hipMalloc(&output, size);
    
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	if(count%2==0)
		sum<<<1,count/2>>>(d,count);
	else
		sum<<<1,(count/2)+1>>>(d,count);
	
	float result;
	hipMemcpy(&result,d, sizeof(float), hipMemcpyDeviceToHost);

  float mean = (float)result/count; 
  
  hipMemcpy(d, h, size, hipMemcpyHostToDevice);

  cout << "\nMean is " << mean << endl;
  
	ss<<<1,count>>>(d,output,mean);
	
  //cudaMemcpy(&sm,output, size, cudaMemcpyDeviceToHost);
  
  //for(int i=0;i<count;i++)
  //  cout<<sm[i]<<" ";
  
  if(count%2==0)
		sum<<<1,count/2>>>(output,count);
	else
		sum<<<1,(count/2)+1>>>(output,count);
	
  float ans;           
	hipMemcpy(&ans,output, sizeof(float), hipMemcpyDeviceToHost);

  ans = (float)ans/count;
  ans = sqrt(ans);  
	cout<<"Standard Deviation is: "<<ans<<endl;
	hipFree(d);

	return 0;
}