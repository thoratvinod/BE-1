#include "hip/hip_runtime.h"
%%cu
#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
using namespace std;

__global__ void max(int* input,int count)
{
	int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{
			int fst = tid * step_size * 2;
			int snd = fst + step_size;
		  if(fst==count-1)
		  {
		      snd = fst;
		      fst = 0;
		  }  
		  if(input[fst]<input[snd])
         	 input[fst] = input[snd]; 
		}

		step_size *= 2; 
		number_of_threads /= 2;
	}
}

int main()
{
	int count = 15;
  srand(time(NULL));
	const int size = count * sizeof(int);
	int h[count];
	for(int i=0;i<count;i++)
    {
		h[i] = rand()%50;
        cout<<h[i]<<" ";
    }
	int* d;
	
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);
 
  if(count%2==0)
	{
	max<<<1,count/2>>>(d,count);
	}else
	{
	max<<<1,(count/2)+1>>>(d,count);
	}
	
	int result;
	hipMemcpy(&result,d, sizeof(int), hipMemcpyDeviceToHost);

    
	cout << "Max is " << result << endl;
	hipFree(d);

	return 0;
}