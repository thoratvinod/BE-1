#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void minElement(int *a, int *s)
{
    atomicMin(s, a[threadIdx.x]);

}

int main()
{
    int num=10, _min;
   
   
    //cout<<"Enter size of array: ";
    //cin>> num;
   
    int a[num];
    //cout<<"Enter array: ";
    for (int i = 0; i < num; i++)
    {
        a[i]=(i*2)+3;
    }
    for (int i = 0; i < num; i++)
    {
        cout << a[i]<<" ";
    }
   
    int *d_a, *d_min;
   
    hipMalloc((void **)&d_a, sizeof(int)*num);
    hipMalloc((void **)&d_min, sizeof(int));    
   
    hipMemcpy(d_a, a, sizeof(int)*num, hipMemcpyHostToDevice);
    int temp = 9999;
    hipMemcpy(d_min, &temp, sizeof(int), hipMemcpyHostToDevice);
   
   
    minElement<<<1, num>>>(d_a, d_min);
   
    hipMemcpy(&_min, d_min, sizeof(int), hipMemcpyDeviceToHost);
   

    hipFree(d_a);
    hipFree(d_min);
   
    cout<<"Minimum elemet: "<<_min<<endl;
   
    return 0;
}
