#include "hip/hip_runtime.h"
%%cu
#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
using namespace std;

__global__ void sum(int* input,int count)
{
	int tid = threadIdx.x;

	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{
			int fst = tid * step_size * 2;
			int snd = fst + step_size;
			if(fst==count-1)
		    	input[0]+=input[fst];
		    else  
				input[fst] += input[snd];
		}
		step_size *= 2; 
		number_of_threads /= 2;
	}
}

int main()
{
	int count = 5;
    int size = count * sizeof(int);
	int h[count];
	for(int i=0;i<count;i++)
    {
		h[i] = rand()%50;
        cout<<h[i]<<" ";
    }
	int* d;
	
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	if(count%2==0)
		sum<<<1,count/2>>>(d,count);
	else
		sum<<<1,(count/2)+1>>>(d,count);
	
	int result;
	hipMemcpy(&result,d, sizeof(int), hipMemcpyDeviceToHost);
  cout << " Sum is: " << result;
    double ans = (double)result/count;  
	cout << " Average is: " << ans << endl;

	hipFree(d);

	return 0;
}