#include "hip/hip_runtime.h"
using namespace std;

__global__ void sum(int *input,int *n)
{
	int tid=threadIdx.x;
	int num_of_threads=blockDim.x;
	float tsize=(float)num_of_threads;

	int step_size=1;
	while(num_of_threads>0)
	{
		if(tid<num_of_threads)
		{
			int fst=tid*step_size*2;
			int snd=fst+step_size;

			if(snd<*n)
			{
				printf("fst = %d %d snd = %d %d\n",fst,input[fst],snd,input[snd]);
				input[fst]+=input[snd];
			}
		}

		step_size*=2;
		if(num_of_threads!=1)
		{
			
			tsize/=2;
			num_of_threads=(int)ceil(tsize);
	
		}
		else
		{
			num_of_threads=0;
		}

	}
}

int main()
{

	int count=5;
	
	//cout<<"Enter number of elements\n";
	//cin>>count;
	int *c=&count;
	const int size=count*sizeof(int);
	
	int a[count];
	for(int i=0;i<count;i++)
	{
		a[i]=rand()%100;
	}

	for(int i=0;i<count;i++)
	{
		cout<<a[i]<<endl;
	}

	int *d;
	int *n;

	hipMalloc(&d,size);
	hipMalloc(&n,sizeof(int));
	hipMemcpy(d,a,size,hipMemcpyHostToDevice);
	hipMemcpy(n,c,sizeof(int),hipMemcpyHostToDevice);
	if(count%2==0)
	{
		sum<<<1,count/2>>>(d,n);
	}else
	{
		sum<<<1,(count/2)+1>>>(d,n);
	}
	int result;
	hipMemcpy(&result,d,sizeof(int),hipMemcpyDeviceToHost);

	cout<<"sum = "<<result<<endl;
	double ans = (double)result/count;  
	cout << " Average is: " << ans << endl;

	hipFree(d);

	return 0;
}
